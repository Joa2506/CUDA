#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//Threads per block
#define BLOCK_SIZE 16

__global__ void gpu_matrixmul(int *a, int *b, int *c, int size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y ;
    int column = blockIdx.x * blockDim.x + threadIdx.x ;

    int sum = 0;
    if((row < size) && (column < size))
    {
        for(int i = 0; i < size; i++)
        {
            sum += a[row * size + i] * b[i * size + column] ;
        }
    }
    c[row * size + column] = sum ;
}

//Linear solution for matrix multiplication.
void matrixmul(int *a, int *b, int *c, int size)
{
    for (int i = 0; i < size; i++) 
    {
        for (int j = 0; j < size; j++)
        {
            for(int k = 0; k < size; k++)
            {
                c[i * size + j] += a[i * size + k] * b[k * size + j];
            }
        }
       // printf("%d\n", i);
    }
}

//Creates two random 2x2 matrices of size.
void create_matrix(int *a, int *b, int size)
{
    int i, j;
    for(i = 0; i < size; i++)
    {
        for (j = 0; j < size; j++)
        {
            a[i * size + j] = rand()%100;
            b[i * size + j] = rand()%100;
        }
    }
    printf("Matrix created!\n");
}

int main()
{

    int size = 1 << 10; //Easy init 1024
    int bytes = size*size*sizeof(int); // size for linear 2x2
    
    //Host
    int *a, *b, *c;

    //GPU
    int *g_a, *g_b, *g_c;


    a = (int*)malloc(bytes);
    b = (int*)malloc(bytes);
    c = (int*)malloc(bytes);

    hipMalloc(&g_a, bytes);
    hipMalloc(&g_b, bytes);
    hipMalloc(&g_c, bytes);

    create_matrix(a, b, size);


    hipMemcpy(g_a, a, bytes,hipMemcpyHostToDevice);
    hipMemcpy(g_b, b, bytes,hipMemcpyHostToDevice);
    //cudaMemcpy(g_c, c, cudaMemcpyHostToDevice);

    //Blocks in each dimension
    int grid_size = (int)ceil(size/BLOCK_SIZE);

    dim3 grid(grid_size, grid_size);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrixmul <<<grid, threads>>>(g_a, g_b, g_c, size);

    hipMemcpy(c, g_c, bytes, hipMemcpyDeviceToHost);

    //matrixmul(a, b, c, size);

    free(a);
    free(b);
    free(c);


}